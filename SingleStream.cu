#include "hip/hip_runtime.h"
 #include <stdio.h>

 //(A+B)/2=C
 
 #define N (2048*2048)//每个流执行数据大小
 #define FULL (N*20)//全部数据大小
 
 __global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < N)
    {
        c[idx] = (a[idx]+b[idx])/2;
    }
}
int main(void)
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("paltform not support overlap");
        return 0;
    }
    //初始化计时器
    hipEvent_t start, stop;
    float elapsedTime;
    //声明流和buffer指针
    hipStream_t stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;
    //开始计时器
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //初始化流
    hipStreamCreate(&stream);
    //GPU端内存申请
    hipMalloc((void **)&dev_a, N*sizeof(int));
    hipMalloc((void **)&dev_b, N*sizeof(int));
    hipMalloc((void **)&dev_c, N*sizeof(int));
    //cpu端分配内存
    hipHostAlloc((void**)&host_a, FULL*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL*sizeof(int), hipHostMallocDefault);
    for(int i =0;i<FULL;i++){
        host_a[i]=rand();
        host_b[i]=rand();
    }
    hipEventRecord(start,0);
    for(int i=0;i<FULL;i+=N){
        //将数据从cpu锁页内存传输给显存
        hipMemcpyAsync(dev_a,host_a+i,N*sizeof(int),hipMemcpyHostToDevice,stream);
        hipMemcpyAsync(dev_b,host_b+i,N*sizeof(int),hipMemcpyHostToDevice,stream);
        kernel<<<N/256,256,0,stream>>>(dev_a,dev_b,dev_c);
        //将计算结果从GPU显存传输给cpu内存
        hipMemcpyAsync(host_c+i,dev_c,N*sizeof(int),hipMemcpyDeviceToHost,stream);
    }
        hipStreamSynchronize(stream);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime,start,stop);
        printf("Time:%3.1f ms\n",elapsedTime);
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        hipHostFree(host_a);
        hipHostFree(host_b);
        hipHostFree(host_c);
        hipStreamDestroy(stream);
    return 0;
}