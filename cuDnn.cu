#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2/opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;

float3 data_kernel[]{
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f),
    make_float3(-2.0f,-2.0f,-2.0f),make_float3(0.0f,0.0f,0.0f),make_float3(2.0f,2.0f,2.0f),
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f),
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f),
    make_float3(-2.0f,-2.0f,-2.0f),make_float3(0.0f,0.0f,0.0f),make_float3(2.0f,2.0f,2.0f),
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f),
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f),
    make_float3(-2.0f,-2.0f,-2.0f),make_float3(0.0f,0.0f,0.0f),make_float3(2.0f,2.0f,2.0f),
    make_float3(-1.0f,-1.0f,-1.0f),make_float3(0.0f,0.0f,0.0f),make_float3(1.0f,1.0f,1.0f)
};

int main(){
    Mat img = imread("1.jpg");
    int imgWidth = img.cols;
    int imgHeight = img.rows;
    imt imgChannel = img.channels();

    Mat dat_gpu(imgHeight, imgWidth, CV_8UC3, Scalar(0,0,0));
    size_t num = imgChannel * imgHeight * imgWidth * sizeof(unsigned char);
    unsigned char *in_gpu;
    unsigned char *out_gpu;
    float *filt_data;
    hipMalloc((viod**)&filt_data, 3*3*3*sizeof(float3));
    hipMalloc((void**)&in_gpu, num);
    hipMalloc((void**)&out_gpu, num);

    hipdnnHandle_t handle;
    hipdnnCreate(&handle);
    // hipdnnSetStream(handle,stream1);
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,1,3,imgHeight,imgWidth);

    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor,HIPDNN_TENSOR_NHWC,HIPDNN_DATA_FLOAT,1,3,imgHeight,imgWidth);

    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    cudnnSetFilter4Descriptor(kernel_descriptor,HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW,3,3,3,3);

    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(conv_descriptor,1,1,1,1,1,1,HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);

    cudnnConvolutionFwdAlgoperf_t algo;
    cudnnGetconvolutionForwardAlgorithm_v7(handle,input_descriptor,kernel_descriptor,
    conv_descriptor,output_descriptor,1,0,&algo);
    
    size_t workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle,input_descriptor,kernel_descriptor,conv_descriptor,output_descriptor,
    algo.algo,&workspace_size);
    void *workspace = nullptr;
    hipMalloc(&workspace,workspace_size);
    hipMemcpy((void*)filt_data,(void*)data_kernel,3*3*3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(in_gpu,img.data,num,hipMemcpyHostToDevice);

    auto alpha = 1.0f,beta=0.0f;
    cudnnGetConvolutionForward(handle,&alpha,input_descriptor,in_gpu,kernel_descriptor,filt_data,conv_descriptor,algo.algo,workspace,workspace_size,&beta,output_descriptor,out_gpu);

    hipMemcpy(dst_gpu.data,out_gpu,num,hipMemcpyDeviceToHost);

    hipFree(in_gpu);
    hipFree(out_gpu);
    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
    hipdnnDestroy(handle);
    return 0;

}